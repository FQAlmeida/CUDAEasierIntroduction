
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

__global__ void add(int n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int n = 1 << 20;
    float *x;
    float *y;

    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));

    for (size_t i = 0; i < n; i++) {
        *(x + i) = 1.0f;
        *(y + i) = 2.0f;
    }
    add<<<1, 256>>>(n, x, y);

    hipDeviceSynchronize();

    float max_error = 0.0f;
    for (int i = 0; i < n; i++)
        max_error = fmax(max_error, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << max_error << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}