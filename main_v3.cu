
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

__global__ void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x +  threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int n = 1 << 20;
    float *x;
    float *y;

    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));

    for (size_t i = 0; i < n; i++) {
        *(x + i) = 1.0f;
        *(y + i) = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(n, x, y);

    hipDeviceSynchronize();

    float max_error = 0.0f;
    for (int i = 0; i < n; i++)
        max_error = fmax(max_error, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << max_error << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}