
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

__global__ void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x +  threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    //printf("threadIdx.x: %d blockIdx.x: %d\n", threadIdx.x, blockIdx.x);
    //printf("threadIdx.y: %d threadIdx.z: %d\n", threadIdx.y, threadIdx.z);
    //printf("blockIdx.y: %d blockIdx.z: %d\n", blockIdx.y, blockIdx.z);
    //printf("blockDim.y: %d blockDim.z: %d\n", blockDim.y, blockDim.z);
    //printf("gridDim.y: %d gridDim.z: %d\n", gridDim.y, gridDim.z);
    for (int i = index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int n = 1 << 20;
    float *x;
    float *y;

    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));

    for (size_t i = 0; i < n; i++) {
        *(x + i) = 1.0f;
        *(y + i) = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(n, x, y);

    hipDeviceSynchronize();

    float max_error = 0.0f;
    for (int i = 0; i < n; i++){
        max_error = fmax(max_error, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << max_error << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}